#include "hip/hip_runtime.h"
#include <cmath>

#include "CudaMain.cuh"
#include "cutil_math.cuh"

#define M_PI 3.14159265359f  // pi

int checkCudaError(hipError_t& error)
{
	if (error == hipSuccess)
	{
		return 0;
	}

	return 1;
}

__global__ void addKernel(float* a, float* b, float* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__device__ inline float3 srgbToLinear(float3 c)
{
	return powf(c, 2.2222f);
}

__device__ inline uint32_t ConvertToRGBA(const float3& color)
{
	float3 outColor;
	outColor.x = clamp(color.x, 0.0f, 1.0f);
	outColor.y = clamp(color.y, 0.0f, 1.0f);
	outColor.z = clamp(color.z, 0.0f, 1.0f);

	outColor = powf(outColor, 0.4646464);

	uint8_t r = (uint8_t)(outColor.x * 255.0f);
	uint8_t g = (uint8_t)(outColor.y * 255.0f);
	uint8_t b = (uint8_t)(outColor.z * 255.0f);

	uint32_t returnValue = (255 << 24) | (b << 16) | (g << 8) | r;

	return returnValue;
}

struct Ray
{
	float3 origin; // ray origin
	float3 direction;  // ray direction
	__device__ Ray(float3 o_, float3 d_) : origin(o_), direction(d_) {}
};

struct Material
{
	float3 albedo    { 0.8f, 0.8f, 0.8f };
	float  roughness { 0.6f };
	float3 emission  { 0.0f, 0.0f, 0.0f };
	float  metalness = 0.0f;
};

struct HitInfo
{
	bool didHit = false;
	float dst = FLT_MAX;
	float3 hitPoint {0.0f, 0.0f, 0.0f};
	float3 normal{ 0.0f, 0.0f, 0.0f };
	size_t materialIndex;
};

struct Sphere
{
	float rad;            // Radius
	float3 pos;           // Position
	size_t materialIndex; // Material Index
};

struct Camera_GPU
{
	float invViewMat[16];
	float invProjMat[16];
	float viewMat[16];
};

void CudaRenderer::Clear()
{
	hipDeviceSynchronize();
	memset(m_outputBuffer, 0, m_bufferSize);
	memset(m_imageData, 0, m_width * m_height * sizeof(uint32_t));
	hipMemset(m_accumulationBuffer_GPU, 0, m_bufferSize);
	hipMemset(m_imageData_GPU, 0, m_width * m_height * sizeof(uint32_t));
}
/*
__constant__ static float jitterMatrix[10] =
{
   -0.25,  0.75,
	0.75,  0.33333,
   -0.75, -0.25,
	0.25, -0.75,
	0.0f, 0.0f
};
*/
// SCENE 9 spheres forming a Cornell box small enough to be in constant GPU memory 
__constant__ Sphere spheres[] =
{
	  { 1e5f,{ 1e5f + 1.0f, 40.8f, 81.6f },     0u }, //Left
	  { 1e5f,{ -1e5f + 99.0f, 40.8f, 81.6f },   1u }, //Right
	  { 1e5f,{ 50.0f, 40.8f, 1e5f },            3u }, //Back
	  { 1e5f,{ 50.0f, 40.8f, -1e5f + 600.0f },  2u }, //Frnt     	   
	  { 1e5f,{ 50.0f, 1e5f, 81.6f },            2u }, //Botm
	  { 1e5f,{ 50.0f, -1e5f + 81.6f, 81.6f },   2u }, //Top			   
	  { 16.5f,{ 27.0f, 16.5f, 47.0f },          2u }, // small sphere 1
	  { 16.5f,{ 73.0f, 16.5f, 78.0f },          4u }, // gold sphere 2
	  { 16.5f,{ 73.0f, 16.5f, 118.0f },         5u }, // copper sphere 2
	  { 100.0f,{ 30.0f, 181.6f - 1.9f, 80.0f }, 6u }, // Light
	  { 100.0f,{ 70.0f, 181.6f - 1.9f, 80.0f }, 7u }  // Light
	  //{ 2.1f,{ 40.0f, 40.5f, 47.0f }, Material{ { 0.8f, 0.8f, 0.8f }, 0.1f, { 150.0f, 160.0f, 180.0f }, 0.0f} }      // Light
};

__constant__  Material materials[] =
{
	Material{ { 0.5f, 0.7f,  0.8f  }, 0.1f, { 0.0f, 0.0f, 0.0f }, 0.0f },	//Blue
	Material{ { 0.7f, 0.1f,  0.1f  }, 0.05f, { 0.0f, 0.0f, 0.0f }, 1.0f },	//Red	
	Material{ { 0.7f, 0.7f,  0.7f  }, 0.05f,{ 0.0f, 0.0f, 0.0f }, 0.0f },   //White
	Material{ { 1.0f, 1.0f,  1.0f  }, 0.0f, { 0.0f, 0.0f, 0.0f }, 1.0f },	//Mirror
	Material{ { 1.0f, 0.9f,  0.6f  }, 0.1f, { 0.0f, 0.0f, 0.0f }, 1.0f },	//Gold
	Material{ { 0.98f,0.815f,0.75f }, 0.1f, { 0.0f, 0.0f, 0.0f }, 1.0f },	//Copper
	Material{ { 0.0f, 0.0f,  0.0f  }, 0.1f, { 8.0f, 6.0f, 5.0f }, 0.0f },	//Light1
	Material{ { 0.0f, 0.0f,  0.0f  }, 0.1f, { 5.0f, 6.0f, 8.0f }, 0.0f }	//Light2
};

__constant__  Sphere spheresSimple[] =
{
	//{ float radius, { float3 position }, { Material }}
	  { 18.0f, { -20.0f, 0.0f, 0.0f }, 1u},
	  { 8.0f, { 0.0f, -10.0f, 0.0f }, 2u},
	  { 6.0f, {  8.0f, 0.0f, 0.0f }, 7u}
};

__device__ static float fresnel_schlick_ratio(float cos_theta_incident, float power)
{
	float p = 1.0f - cos_theta_incident;
	return pow(p, power);
}
__constant__ static float jitterMatrix[10] =
{
   -0.25,  0.75,
	0.75,  0.33333,
   -0.75, -0.25,
	0.25, -0.75,
	0.0f, 0.0f
};

// PCG (permuted congruential generator). Thanks to:
// www.pcg-random.org and www.shadertoy.com/view/XlGcRh
__device__ uint32_t nextRandom(uint32_t& state)
{
	state = state * 747796405 + 2891336453;
	uint result = ((state >> ((state >> 28) + 4)) ^ state) * 277803737;
	result = (result >> 22) ^ result;
	return result;
}

__device__ float randomValue(uint32_t& state)
{
	return nextRandom(state) / 4294967295.0; // 2^32 - 1
}

__device__ float3 inUnitSphere(uint32_t& state)
{
	return normalize(make_float3(randomValue(state) * 2.0f - 1.0f, randomValue(state) * 2.0f - 1.0f, randomValue(state) * 2.0f - 1.0f));
}

__device__ float randomValueNormalDistribution(uint32_t& state)
{
	// Thanks to https://stackoverflow.com/a/6178290
	float theta = 2 * 3.1415926 * randomValue(state);
	float rho = fsqrtf(-2 * log(randomValue(state)));

	return rho * cos(theta);
}

__device__ float3 randomDirection(uint32_t& state)
{
	// Thanks to https://math.stackexchange.com/a/1585996
	float x = randomValueNormalDistribution(state);
	float y = randomValueNormalDistribution(state);
	float z = randomValueNormalDistribution(state);

	return normalize(make_float3(x, y, z));
}

__device__ void vector4_matrix4_mult(float* vec, float* mat, float* out)
{
	for (int i = 0; i < 4; i++)
	{
		out[i] = 0.0f;
	}

	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			out[i] += (mat[i + 4 * j] * vec[j]);
		}
	}
}

__device__ float3 getEnvironmentLight(const Ray& ray)
{
	float3 sunDir = make_float3(1.0f, 1.0f, 1.0f);
	sunDir = normalize(sunDir);

	float skyGradientT = powf(fmaxf(ray.direction.y, 0.0f), 0.35f);
	float groundToSkyT = powf(fmaxf(ray.direction.y, 0.0f), 0.1f);

	float3 skyColorHorizon{ 0.308, 0.459, 0.670 };
	float3 skyColorZenith{ 0.0416, 0.158, 0.320 };
	float3 groundColor{ 0.110, 0.102, 0.0891 };

	float3 skyGradient = lerp(skyColorHorizon, skyColorZenith, skyGradientT);
	float sun = powf(fmaxf(0.0f, dot(ray.direction, sunDir)), 100.0f) * 20.0f;

	// Combine ground, sky, and sun
	float3 composite = lerp(groundColor, skyGradient, groundToSkyT) + sun;

	return composite;
}

__device__ HitInfo intersect_sphere(const Ray& r, const Sphere& s)
{
	HitInfo hit;

	float3 offsetRayOrigin = r.origin - s.pos;
	float a = dot(r.direction, r.direction);
	float b = 2.0f * dot(offsetRayOrigin, r.direction);
	float c = dot(offsetRayOrigin, offsetRayOrigin) - s.rad * s.rad;
	// Quadratic discriminant
	float discriminant = b * b - 4.0f * a * c;

	// No solution when d < 0 (ray misses sphere)
	if (discriminant >= 0.0f)
	{
		// Distance to nearest intersection point (from quadratic formula)
		float dst = (-b - fsqrtf(discriminant)) / (2.0f * a);

		// Ignore intersections that occur behind the ray
		if (dst >= 0.0f)
		{
			hit.didHit = true;
			hit.dst = dst;
			hit.hitPoint = r.origin + r.direction * dst;
			hit.normal = normalize(hit.hitPoint - s.pos);
			hit.materialIndex = s.materialIndex;
		}
	}

	return hit;
}

__device__ HitInfo rayTriangleIntersect(const Ray& ray, const float3& v0, const float3& v1, const float3& v2, const float3& vn0, const float3& vn1, const float3& vn2)
{
	float3 edgeAB = v1 - v0;
	float3 edgeAC = v2 - v0;
	float3 normalVector = cross(edgeAB, edgeAC);
	float3 ao = (ray.origin) - v0;
	float3 dao = cross(ao, ray.direction);

	float determinant = -dot(ray.direction, normalVector);
	float invDet = 1.0f / determinant;

	// Calculate dst to triangle & barycentric coordinates of intersection point
	float dst = dot(ao, normalVector) * invDet;
	float u = dot(edgeAC, dao) * invDet;
	float v = -dot(edgeAB, dao) * invDet;
	float w = 1.0f - u - v;

	// Initialize hit info
	HitInfo hit;
	hit.didHit = determinant >= 1E-6 && dst >= 0.0f && u >= 0.0f && v >= 0.0f && w >= 0.0f;
	hit.hitPoint = (ray.origin) + ray.direction * dst;
	hit.normal = normalize(vn0 * w + vn1 * u + vn2 * v);
	//hit.normal = normalVector;
	hit.dst = dst;

	return hit;
}

__device__ bool rayBoundingBox(const Ray& ray, const float3& min, float3& max)
{
	float3 invDir = 1.0f / ray.direction;
	float3 tMin = (min - ray.origin) * invDir;
	float3 tMax = (max - ray.origin) * invDir;
	float3 t1 = fminf(tMin, tMax);
	float3 t2 = fmaxf(tMin, tMax);

	float tNear = fmaxf(fmaxf(t1.x, t1.y), t1.z);
	float tFar  = fminf(fminf(t2.x, t2.y), t2.z);

	return tNear <= tFar;
}

__device__ HitInfo intersect_triangles(const Ray& r, const GPU_Mesh::GPU_MeshList* vbo)
{
	HitInfo hit;
	HitInfo closestHit;

	if (rayBoundingBox(r, vbo->bboxMins[0], vbo->bboxMaxs[0]))
	{
		closestHit.didHit = false;
		return closestHit;
	}

	size_t n = vbo->vertexCounts[0] * vbo->vertexStride;     // t is distance to closest intersection, initialise t to a huge number outside scene
	for (size_t i = 0; i < n; i += 24u)                                      // Test all scene objects for intersection
	{
		float3 vp0 = make_float3(vbo->vertexBuffer[i], vbo->vertexBuffer[i + 1], vbo->vertexBuffer[i + 2]);
		float3 vp1 = make_float3(vbo->vertexBuffer[i+8], vbo->vertexBuffer[i+9], vbo->vertexBuffer[i+10]);
		float3 vp2 = make_float3(vbo->vertexBuffer[i+16], vbo->vertexBuffer[i+17], vbo->vertexBuffer[i+18]);

		float3 vn0 = make_float3(vbo->vertexBuffer[i + 3], vbo->vertexBuffer[i + 4], vbo->vertexBuffer[i + 5]);
		float3 vn1 = make_float3(vbo->vertexBuffer[i + 11], vbo->vertexBuffer[i + 12], vbo->vertexBuffer[i + 13]);
		float3 vn2 = make_float3(vbo->vertexBuffer[i + 19], vbo->vertexBuffer[i + 20], vbo->vertexBuffer[i + 21]);


		hit = rayTriangleIntersect(r, vp0, vp1, vp2, vn0, vn1, vn2);

		if (hit.didHit && hit.dst < closestHit.dst) // If newly computed intersection distance d is smaller than current closest intersection distance
		{
			closestHit = hit;
		}
	}
	// Returns true if an intersection with the scene occurred, false when no hit
	return closestHit;
}

__device__ HitInfo intersect_scene(const Ray& r, const GPU_Mesh::GPU_MeshList* vbo)
{
	HitInfo hit;
	HitInfo closestHit;

	float n = sizeof(spheresSimple) / sizeof(Sphere);

	for (size_t i = 0u; i < size_t(n); i++)
	{
		Sphere s = spheresSimple[i];
		hit = intersect_sphere(r, s);

		if (hit.didHit && hit.dst < closestHit.dst) // If newly computed intersection distance d is smaller than current closest intersection distance
		{
			closestHit = hit;
		}
	}

	hit = intersect_triangles(r, vbo);

	if (hit.didHit && hit.dst < closestHit.dst)
	{
		closestHit = hit;
		closestHit.materialIndex = 2u;
	}

	// Returns true if an intersection with the scene occurred, false when no hit
	return closestHit;
}

__device__ float3 radianceTris(Ray& r, uint32_t& s1, size_t bounces, const GPU_Mesh::GPU_MeshList* vbo) // Returns ray color
{
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f); // Accumulates ray colour with each iteration through bounce loop
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);

	for (size_t b = 0; b < bounces; b++)
	{
		// Test ray for intersection with scene
		HitInfo hit = intersect_scene(r, vbo);
		if (!hit.didHit)
		{
			//accucolor += mask * make_float3(0.0494, 0.091, 0.164f); // If miss, return sky
			//accucolor += mask * getEnvironmentLight(r) * 0.0f;
			break;
		}

		Material hitMat = materials[hit.materialIndex];

		accucolor += mask * hitMat.emission;

		// Create 2 random numbers
		float r1 = 2 * M_PI * randomValue(s1); // Pick random number on unit circle (radius = 1, circumference = 2*Pi) for azimuth
		float r2 = randomValue(s1);            // Pick random number for elevation
		float r2s = sqrtf(r2);

		float ndotl = fmaxf(dot(-r.direction, hit.normal), 0.0f);
		float f = fresnel_schlick_ratio(ndotl, 8.0f);

		bool isSpecularBounce = max(hitMat.metalness, max(f, 0.02f)) >= randomValue(s1);

		float3 diffuseDir = normalize(hit.normal + randomDirection(s1));
		float3 specularDir = reflect(r.direction, normalize(hit.normal + randomDirection(s1) * hitMat.roughness));

		float3 linearSurfColor = powf(hitMat.albedo, 2.2f);

		r.direction = normalize(lerp(diffuseDir, specularDir, isSpecularBounce));
		r.origin = hit.hitPoint + hit.normal * 0.001f; // offset ray origin slightly to prevent self intersection

		mask = mask * lerp(linearSurfColor, lerp(make_float3(1.0f), linearSurfColor, hitMat.metalness), isSpecularBounce);

		//float p = max(mask.x, max(mask.y, mask.z));
		//if (randomValue(s1) >= p)
		{
		//	break;
		}
		//mask *= 1.0f / p;

		//accucolor = { hit.normal };
	}


	return accucolor;
}

/*
__device__ float3 radiance(Ray& r, uint32_t& s1, size_t bounces) // Returns ray color
{
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f); // Accumulates ray colour with each iteration through bounce loop
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);

	for (size_t b = 0; b < bounces; b++)
	{
		float t;           // Distance to closest intersection
		int id = 0;        // Index of closest intersected sphere

		// Test ray for intersection with scene
		if (!intersect_scene(r))
		{
			accucolor += mask * make_float3(0.1f, 0.12f, 0.2f); // If miss, return sky
			break;
		}
		const Sphere& obj = spheres[id];
		float3 x = r.origin + r.direction * t;                   // hitpoint
		float3 n = normalize(x - obj.pos);             // normal
		float3 nl = dot(n, r.direction) < 0.0f ? n : n * -1.0f;    // front facing normal

		accucolor += mask * obj.mat.emission;

		// Create 2 random numbers
		float r1 = 2 * M_PI * randomValue(s1); // Pick random number on unit circle (radius = 1, circumference = 2*Pi) for azimuth
		float r2 = randomValue(s1);            // Pick random number for elevation
		float r2s = sqrtf(r2);

		float ndotl = fmaxf(dot(-r.direction, nl), 0.0f);
		float f = fresnel_schlick_ratio(ndotl, 8.0f);

		bool isSpecularBounce = max(obj.mat.metalness, max(f, 0.02f)) >= randomValue(s1);

		float3 diffuseDir = normalize(nl + randomDirection(s1));
		float3 specularDir = reflect(r.direction, normalize(nl + randomDirection(s1) * obj.mat.roughness));

		float3 linearSurfColor = srgbToLinear(obj.mat.albedo);

		r.direction = normalize(lerp(diffuseDir, specularDir, isSpecularBounce));

		// New ray origin is intersection point of previous ray with scene
		r.origin = x + nl * 0.1f; // offset ray origin slightly to prevent self intersection

		mask = mask * lerp(linearSurfColor, lerp(make_float3(1.0f), linearSurfColor, obj.mat.metalness), isSpecularBounce);

		float p = max(mask.x, max(mask.y, mask.z));
		if (randomValue(s1) >= p)
		{
			break;
		}
		mask *= 1.0f / p;

		//accucolor = { f, f, f };
	}


	return accucolor;
}
*/

__global__ void render_kernel(float3* buf, uint32_t width, uint32_t height, Camera_GPU camera, size_t samples, size_t bounces, uint32_t sampleIndex, const GPU_Mesh::GPU_MeshList* vbo)
{
	// Assign a CUDA thread to every pixel (x,y) blockIdx, blockDim and threadIdx are CUDA specific
	// Keywords replaces nested outer loops in CPU code looping over image rows and image columns
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= width) || (y >= height)) return;

	// Index of current pixel (calculated using thread index)
	uint32_t i = (height - y - 1) * width + x;
	
	// Seeds for random number generator
	uint32_t s1 = x * y * sampleIndex + i;

	float2 coord = { (float)x / (float)width, (float)y / (float)height };
	coord = coord * 2.0f - make_float2(1.0f, 1.0f); // -1 -> 1
	float viewCoord[4] = { coord.x, coord.y, -1.0f, 1.0f };
	float target[4];
	float target2[4];

	vector4_matrix4_mult(&viewCoord[0], &camera.invProjMat[0], &target[0]);

	//float4 target = m_InverseProjection * ;
	float4 projDir4 = make_float4(normalize(make_float3(target[0], target[1], target[2]) / target[3]), 0.0f);

	float projDir[4] = { projDir4.x, projDir4.y, projDir4.z, projDir4.w };

	vector4_matrix4_mult(&projDir[0], &camera.invViewMat[0], target2);

	float3 worldDir = normalize(make_float3(target2[0], target2[1], target2[2]));

	float3 cx = make_float3(camera.invViewMat[0], camera.invViewMat[1], camera.invViewMat[2]);
	float3 cy = make_float3(camera.invViewMat[4], camera.invViewMat[5], camera.invViewMat[6]);
	float3 cz = make_float3(camera.invViewMat[8], camera.invViewMat[9], camera.invViewMat[10]);

	float3 lightContribution;

	// Reset r to zero for every pixel
	lightContribution = make_float3(0.0f);

	float3 cameraPos = make_float3(camera.invViewMat[12], camera.invViewMat[13], camera.invViewMat[14]);

	// Samples per pixel
	for (size_t s = 0; s < samples; s++)
	{
		size_t jitterIndex = (s + sampleIndex) % 5u;
		float jitterX = (jitterMatrix[2u * jitterIndex]);
		float jitterY = (jitterMatrix[2u * jitterIndex + 1u]);

		// Compute primary ray direction
		float3 d = (cx * (jitterX / width) + cy * (jitterY / height));

		// Create primary ray, add incoming radiance to pixelcolor
		Ray ray = Ray(cameraPos, normalize(worldDir + d*0.5f));
		lightContribution += radianceTris(ray, s1, bounces, vbo) * (1.0 / samples);
	}

	// Write rgb value of pixel to image buffer on the GPU
	buf[i] += lightContribution;
}

__global__ void floatToImageData_kernel(uint32_t* outputBuffer, float3* inputBuffer, uint32_t width, uint32_t height, uint32_t sampleIndex)
{
	uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
				 

	if ((x >= width) || (y >= height))
		return;

	// Index of current pixel (calculated using thread index)
	uint32_t i = (height - y - 1) * width + x;

	outputBuffer[i] = ConvertToRGBA(inputBuffer[i] / (float)sampleIndex);
}

// Initialize and run the kernel
void CudaRenderer::Compute(void)
{
	int tx = 8;
	int ty = 8;

	// dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
	dim3 blocks(m_width / tx + 1, m_height / ty + 1, 1);
	dim3 threads(tx, ty);

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	if (m_gpuMesh->hasChanged || deviceStruct == nullptr)
	{
		cudaStatus = hipMalloc(&deviceStruct, sizeof(GPU_Mesh::GPU_MeshList));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(deviceStruct, m_meshList, sizeof(GPU_Mesh::GPU_MeshList), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		float* d_vbo;
		hipMalloc(&d_vbo, m_meshList->meshOffsets[0]);
		hipMemcpy(d_vbo, m_meshList->vertexBuffer, m_meshList->meshOffsets[0], hipMemcpyHostToDevice);
		hipMemcpy(&deviceStruct->vertexBuffer, &d_vbo, sizeof(float*), hipMemcpyHostToDevice);

		size_t* d_meshOffsets;
		hipMalloc(&d_meshOffsets, m_meshList->meshCount * sizeof(size_t));
		hipMemcpy(d_meshOffsets, m_meshList->meshOffsets, m_meshList->meshCount * sizeof(size_t), hipMemcpyHostToDevice);
		hipMemcpy(&deviceStruct->meshOffsets, &d_meshOffsets, sizeof(size_t*), hipMemcpyHostToDevice);

		size_t* d_vertexCounts;
		hipMalloc(&d_vertexCounts, m_meshList->meshCount * sizeof(size_t));
		hipMemcpy(d_vertexCounts, m_meshList->vertexCounts, m_meshList->meshCount * sizeof(size_t), hipMemcpyHostToDevice);
		hipMemcpy(&deviceStruct->vertexCounts, &d_vertexCounts, sizeof(size_t*), hipMemcpyHostToDevice);

		float3* d_bboxMin;
		hipMalloc(&d_bboxMin, m_meshList->meshCount * sizeof(float3));
		hipMemcpy(d_bboxMin, &m_meshList->bboxMins[0], m_meshList->meshCount * sizeof(float3), hipMemcpyHostToDevice);
		hipMemcpy(&deviceStruct->bboxMins, &d_bboxMin, sizeof(float3*), hipMemcpyHostToDevice);

		float3* d_bboxMax;
		hipMalloc(&d_bboxMax, m_meshList->meshCount * sizeof(float3));
		hipMemcpy(d_bboxMax, &m_meshList->bboxMins[0], m_meshList->meshCount * sizeof(float3), hipMemcpyHostToDevice);
		hipMemcpy(&deviceStruct->bboxMaxs, &d_bboxMax, sizeof(float3*), hipMemcpyHostToDevice);

		m_gpuMesh->hasChanged = false;
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	Camera_GPU camera_buffer_obj;
	memcpy(&camera_buffer_obj.invProjMat[0], m_invProjMat, sizeof(float) * 16);
	memcpy(&camera_buffer_obj.invViewMat[0], m_invViewMat, sizeof(float) * 16);
	memcpy(&camera_buffer_obj.viewMat[0],    m_viewMat,    sizeof(float) * 16);

	render_kernel <<<blocks, threads>>> (m_accumulationBuffer_GPU, m_width, m_height, camera_buffer_obj, m_samples, *m_bounces, *m_sampleIndex, deviceStruct);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "render_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipDeviceSynchronize();

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching render_kernel!\n", cudaStatus);
		goto Error;
	}

	floatToImageData_kernel <<<blocks, threads >>> (m_imageData_GPU, m_accumulationBuffer_GPU, m_width, m_height, *m_sampleIndex);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "floatToImageData_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching floatToImageData_kernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(m_imageData, m_imageData_GPU, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//hipFree(output_buffer_gpu);

	Error:
}

void CudaRenderer::SetCamera(float3 pos, float3 dir, float fov)
{
	m_cameraPos = pos;
	m_cameraDir = dir;
	m_fov = fov;
}

void CudaRenderer::SetInvViewMat(float4 x, float4 y, float4 z, float4 w)
{
	m_invViewMat[0]  = x.x;
	m_invViewMat[1]  = x.y;
	m_invViewMat[2]  = x.z;
	m_invViewMat[3]  = x.w;
				    
	m_invViewMat[4]  = y.x;
	m_invViewMat[5]  = y.y;
	m_invViewMat[6]  = y.z;
	m_invViewMat[7]  = y.w;

	m_invViewMat[8]  = z.x;
	m_invViewMat[9]  = z.y;
	m_invViewMat[10] = z.z;
	m_invViewMat[11] = z.w;

	m_invViewMat[12] = w.x;
	m_invViewMat[13] = w.y;
	m_invViewMat[14] = w.z;
	m_invViewMat[15] = w.w;
}

void CudaRenderer::SetInvProjMat(float4 x, float4 y, float4 z, float4 w)
{
	m_invProjMat[0] = x.x;
	m_invProjMat[1] = x.y;
	m_invProjMat[2] = x.z;
	m_invProjMat[3] = x.w;

	m_invProjMat[4] = y.x;
	m_invProjMat[5] = y.y;
	m_invProjMat[6] = y.z;
	m_invProjMat[7] = y.w;

	m_invProjMat[8] = z.x;
	m_invProjMat[9] = z.y;
	m_invProjMat[10] = z.z;
	m_invProjMat[11] = z.w;

	m_invProjMat[12] = w.x;
	m_invProjMat[13] = w.y;
	m_invProjMat[14] = w.z;
	m_invProjMat[15] = w.w;
}

void CudaRenderer::SetViewMat(float4 x, float4 y, float4 z, float4 w)
{
	m_viewMat[0] = x.x;
	m_viewMat[1] = x.y;
	m_viewMat[2] = x.z;
	m_viewMat[3] = x.w;

	m_viewMat[4] = y.x;
	m_viewMat[5] = y.y;
	m_viewMat[6] = y.z;
	m_viewMat[7] = y.w;

	m_viewMat[8] = z.x;
	m_viewMat[9] = z.y;
	m_viewMat[10] = z.z;
	m_viewMat[11] = z.w;

	m_viewMat[12] = w.x;
	m_viewMat[13] = w.y;
	m_viewMat[14] = w.z;
	m_viewMat[15] = w.w;
}