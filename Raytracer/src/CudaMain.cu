#include "hip/hip_runtime.h"
#include "CudaMain.cuh"

int checkCudaError(hipError_t& error)
{
	if (error == hipSuccess)
	{
		return 0;
	}

	return 1;
}

__global__ void addKernel(float* a, float* b, float* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float* a, float* b, float* c, unsigned int size)
{
	float* dev_a = 0;
	float* dev_b = 0;
	float* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (checkCudaError(cudaStatus)) { goto Error; }

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	if (checkCudaError(cudaStatus)) { goto Error; }

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	if (checkCudaError(cudaStatus)) { goto Error; }

	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
	if (checkCudaError(cudaStatus)) { goto Error; }

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	if (checkCudaError(cudaStatus)) { goto Error; }

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
	if (checkCudaError(cudaStatus)) { goto Error; }

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size>>> (dev_a, dev_b, dev_c);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (checkCudaError(cudaStatus)) { goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (checkCudaError(cudaStatus)) { goto Error; }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	if (checkCudaError(cudaStatus)) { goto Error; }

Error:
	//fprintf(stderr, "Cuda kernel failed!");
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

void CudaBuffer::Compute(void)
{
	hipError_t cudaStatus = addWithCuda(m_cudaBufferA, m_cudaBufferB, m_cudaBufferC, (unsigned int)m_bufferSize);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Cuda compute failed!");
	}
}